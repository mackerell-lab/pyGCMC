#include "hip/hip_runtime.h"
/*
    © Copyright 2023 - University of Maryland, Baltimore   All Rights Reserved
        Mingtian Zhao, Alexander D. MacKerell Jr.
    E-mail:
        zhaomt@outerbanks.umaryland.edu
        alex@outerbanks.umaryland.edu
*/

#include "gcmc.h"
#include "gcmc_move.h"

extern "C"{

    // Initialize random number generator states for CUDA threads
    __global__ void setup_rng_states(hiprandState *states, unsigned long long seed) {
        int global_threadIdx  = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, global_threadIdx, 0, &states[global_threadIdx]);
    }

    // Main GCMC simulation function
    void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){
        
        InfoStruct *Ginfo;
        AtomArray *GfragmentInfo;
        residue *GresidueInfo; 
        Atom *GatomInfo;
        float *Ggrid;
        float *Gff;
        int *GmoveArray;

        hipMalloc(&Ginfo, sizeof(InfoStruct));
        hipMalloc(&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
        hipMalloc(&GresidueInfo, sizeof(residue)*info->totalResNum);
        hipMalloc(&GatomInfo, sizeof(Atom)*info->totalAtomNum);
        hipMalloc(&Ggrid, sizeof(float)*info->totalGridNum * 3);
        hipMalloc(&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);

        hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
        hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
        hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice) ;
        hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
        hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
        hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);

        int maxConf = 0;
        for (int fragType = 0; fragType < info->fragTypeNum; fragType ++ ){
            if (fragmentInfo[fragType].confBias > maxConf){
                maxConf = fragmentInfo[fragType].confBias;
            }
        }

        AtomArray *GTempFrag;
        hipMalloc(&GTempFrag, sizeof(AtomArray)*maxConf);

        Atom *GTempInfo;
        hipMalloc(&GTempInfo, sizeof(Atom)*maxConf);

        Atom *TempInfo;
        TempInfo = (Atom *)malloc(sizeof(Atom)*maxConf);

        for (int i = 0;i < maxConf; i++){
            TempInfo[i].type = 0;
        }

        hipMemcpy(GTempInfo, TempInfo, sizeof(Atom)*maxConf, hipMemcpyHostToDevice);

        hiprandState *d_rng_states;
        
        hipMalloc((void **)&d_rng_states, maxConf * sizeof(hiprandState) * numThreadsPerBlock);

        srand(info->seed);

        setup_rng_states<<<maxConf, numThreadsPerBlock>>>(d_rng_states, info->seed);

        int step_threshold = info->mcsteps / 20;

        for (int stepi = 0 ; stepi < info->mcsteps; ++stepi){
            int moveFragType = moveArray[stepi] / 4;
            int moveMoveType = moveArray[stepi] % 4;
            int confBias = fragmentInfo[moveFragType].confBias;

            bool accepted = false;
            switch (moveMoveType)
            {
            case 0: // Insert
                accepted = move_add(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;
            case 1: // Delete
                accepted = move_del(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;
            case 2: // Translate
                accepted = move_trn(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;
            case 3: // Rotate
                accepted = move_rot(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;
            }
        }
        printf("\n");

        hipDeviceSynchronize();

        hipMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyDeviceToHost);
        hipMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, hipMemcpyDeviceToHost);
        hipMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyDeviceToHost);

        hipFree(Ginfo);
        hipFree(GfragmentInfo);
        hipFree(GresidueInfo);
        hipFree(GatomInfo);
        hipFree(Ggrid);
        hipFree(Gff);
        hipFree(GTempFrag);
        hipFree(GTempInfo);
        hipFree(d_rng_states);

        free(TempInfo);
    }
}


