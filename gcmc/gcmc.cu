#include "hip/hip_runtime.h"
/*
    © Copyright 2023 - University of Maryland, Baltimore   All Rights Reserved
        Mingtian Zhao, Abhishek A. Kognole,
        Aoxiang Tao, Alexander D. MacKerell Jr.
    E-mail:
        zhaomt@outerbanks.umaryland.edu
        alex@outerbanks.umaryland.edu
*/


#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

extern "C" {


    __global__ void vector_add_kernel(const float *a, const float *b, float *c, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N) {
            c[i] = a[i] + b[i];
        }
    }

    void vector_add_cuda(const float *a, const float *b, float *c, int N) {
        float *d_a, *d_b, *d_c;

        hipMalloc(&d_a, N * sizeof(float));
        hipMalloc(&d_b, N * sizeof(float));
        hipMalloc(&d_c, N * sizeof(float));

        hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        vector_add_kernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

        // Synchronize device
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    __global__ void vector_sub_kernel(const float *a, const float *b, float *c, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N) {
            c[i] = a[i] - b[i];
        }
    }

    void vector_sub_cuda(const float *a, const float *b, float *c, int N) {
        float *d_a, *d_b, *d_c;

        hipMalloc(&d_a, N * sizeof(float));
        hipMalloc(&d_b, N * sizeof(float));
        hipMalloc(&d_c, N * sizeof(float));

        hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;
        vector_sub_kernel<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

        // Synchronize device
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

