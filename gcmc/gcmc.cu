/*
    © Copyright 2023 - University of Maryland, Baltimore   All Rights Reserved
        Mingtian Zhao, Abhishek A. Kognole,
        Aoxiang Tao, Alexander D. MacKerell Jr.
    E-mail:
        zhaomt@outerbanks.umaryland.edu
        alex@outerbanks.umaryland.edu
*/


#include <hip/hip_runtime.h>
// #include <unistd.h>
// #include <thrust/device_vector.h>
#include "gcmc.h"

// #include <cstdio>
// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }

extern "C"{
    // void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){}
    void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){
        
        InfoStruct *Ginfo;
        AtomArray *GfragmentInfo;
        residue *GresidueInfo; 
        Atom *GatomInfo;
        float *Ggrid;
        float *Gff;
        int *GmoveArray;

        hipMalloc(&Ginfo, sizeof(InfoStruct));
        hipMalloc(&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
        hipMalloc(&GresidueInfo, sizeof(residue)*info->totalResNum);
        hipMalloc(&GatomInfo, sizeof(Atom)*info->totalAtomNum);
        hipMalloc(&Ggrid, sizeof(float)*info->totalGridNum * 3);
        hipMalloc(&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
        // hipMalloc(&GmoveArray, sizeof(int)*info->mcsteps);

        // printf("hipMalloc done\n");

        // sleep(360);

        hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
        // printf("hipMemcpy Ginfo done\n");
        hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
        // printf("hipMemcpy GfragmentInfo done\n");
        hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice) ;
        // printf("hipMemcpy GresidueInfo done\n");
        hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
        // printf("hipMemcpy GatomInfo done\n");
        hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
        // printf("hipMemcpy Ggrid done\n");
        hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);
        // printf("hipMemcpy Gff done\n");
        // hipMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, hipMemcpyHostToDevice);
        // printf("hipMemcpy GmoveArray done\n");

        for (int stepi = 0 ; stepi < info->mcsteps; ++stepi){
            // Start MC steps
            int moveFragType = moveArray[stepi] / 4;
            int moveMoveType = moveArray[stepi] % 4;
            int confBias = fragmentInfo[moveFragType].confBias;

            // // perform move
            // bool accepted = false;
            // switch (moveMoveType)
            // {
            // case 0: // Insert
            //     accepted = move_add(Ginfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, confBias);
            //     break;

            // case 1: // Del
            //     accepted = move_del(frag_index);
            //     break;

            // case 2: // Trn
            //     accepted = move_trans(frag_index);
            //     break;

            // case 3: // Rot
            //     accepted = move_rotate(frag_index);
            //     break;
            // }






        }




        hipDeviceSynchronize();


        hipMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyDeviceToHost);
        hipMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, hipMemcpyDeviceToHost);
        hipMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyDeviceToHost);

        hipFree(Ginfo);
        hipFree(GfragmentInfo);
        hipFree(GresidueInfo);
        hipFree(GatomInfo);
        hipFree(Ggrid);
        hipFree(Gff);
        // hipFree(GmoveArray);
        
    }
}

// extern "C" {


//     void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){


//         // InfoStruct *Ginfo;
//         // AtomArray *GfragmentInfo;
//         // residue *GresidueInfo; 
//         // Atom *GatomInfo;
//         // float *Ggrid;
//         // float *Gff;
//         // int *GmoveArray;


//         // hipMalloc(&Ginfo, sizeof(InfoStruct));

//         // hipMalloc((void**)&Ginfo, sizeof(InfoStruct));
//         // hipMalloc((void**)&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
//         // hipMalloc((void**)&GresidueInfo, sizeof(residue)*info->totalResNum);
//         // hipMalloc((void**)&GatomInfo, sizeof(Atom)*info->totalAtomNum);
//         // hipMalloc((void**)&Ggrid, sizeof(float)*info->totalGridNum * 3);
//         // hipMalloc((void**)&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
//         // hipMalloc((void**)&GmoveArray, sizeof(int)*info->mcsteps);

//         // hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
//         // hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
//         // hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice);
//         // hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
//         // hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
//         // hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);
//         // hipMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, hipMemcpyHostToDevice);

        
//         // hipDeviceSynchronize();

//         // sleep(60);

//         // hipMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyDeviceToHost);
//         // hipMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, hipMemcpyDeviceToHost);
//         // hipMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyDeviceToHost);

//         // hipFree(Ginfo);
//         // hipFree(GfragmentInfo);
//         // hipFree(GresidueInfo);
//         // hipFree(GatomInfo);
//         // hipFree(Ggrid);
//         // hipFree(Gff);
//         // hipFree(GmoveArray);

//     }



// }

