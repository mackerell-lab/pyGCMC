/*
    © Copyright 2023 - University of Maryland, Baltimore   All Rights Reserved
        Mingtian Zhao, Abhishek A. Kognole,
        Aoxiang Tao, Alexander D. MacKerell Jr.
    E-mail:
        zhaomt@outerbanks.umaryland.edu
        alex@outerbanks.umaryland.edu
*/


#include <iostream>
#include <hip/hip_runtime.h>
#include <unistd.h>
// #include <thrust/device_vector.h>

struct Atom {
    float position[3];
    float charge;
    int type;
};

struct AtomArray {
    
    char name[4];

    int startRes;

    float muex;
    float conc;
    float confBias;
    float mcTime;
    
    int totalNum;
    int maxNum;

    int num_atoms;
    Atom atoms[20];
};

struct InfoStruct{
    int mcsteps;
    float cutoff;
    float grid_dx;
    float startxyz[3];
    float cryst[3];

    float cavityFactor;
    
    int fragTypeNum;
    
    int totalGridNum;
    int totalResNum;
    int totalAtomNum;
    
    int ffXNum;
    int ffYNum;
};

struct residue{
    float position[3];
    int atomNum;
    int atomStart;
};


extern "C"{
    // void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){}
    void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){
        
        InfoStruct *Ginfo;
        AtomArray *GfragmentInfo;
        residue *GresidueInfo; 
        Atom *GatomInfo;
        float *Ggrid;
        float *Gff;
        int *GmoveArray;

        hipMalloc(&Ginfo, sizeof(InfoStruct));
        hipMalloc(&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
        hipMalloc(&GresidueInfo, sizeof(residue)*info->totalResNum);
        hipMalloc(&GatomInfo, sizeof(Atom)*info->totalAtomNum);
        hipMalloc(&Ggrid, sizeof(float)*info->totalGridNum * 3);
        hipMalloc(&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
        hipMalloc(&GmoveArray, sizeof(int)*info->mcsteps);

        sleep(60);

        hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
        hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
        hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice);
        hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
        hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
        hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);
        hipMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, hipMemcpyHostToDevice);

        
    }
}

// extern "C" {


//     void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){


//         // InfoStruct *Ginfo;
//         // AtomArray *GfragmentInfo;
//         // residue *GresidueInfo; 
//         // Atom *GatomInfo;
//         // float *Ggrid;
//         // float *Gff;
//         // int *GmoveArray;


//         // cudaMalloc(&Ginfo, sizeof(InfoStruct));

//         // cudaMalloc((void**)&Ginfo, sizeof(InfoStruct));
//         // cudaMalloc((void**)&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
//         // cudaMalloc((void**)&GresidueInfo, sizeof(residue)*info->totalResNum);
//         // cudaMalloc((void**)&GatomInfo, sizeof(Atom)*info->totalAtomNum);
//         // cudaMalloc((void**)&Ggrid, sizeof(float)*info->totalGridNum * 3);
//         // cudaMalloc((void**)&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
//         // cudaMalloc((void**)&GmoveArray, sizeof(int)*info->mcsteps);

//         // cudaMemcpy(Ginfo, info, sizeof(InfoStruct), cudaMemcpyHostToDevice);
//         // cudaMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, cudaMemcpyHostToDevice);
//         // cudaMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, cudaMemcpyHostToDevice);
//         // cudaMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, cudaMemcpyHostToDevice);
//         // cudaMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, cudaMemcpyHostToDevice);
//         // cudaMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, cudaMemcpyHostToDevice);
//         // cudaMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, cudaMemcpyHostToDevice);

        
//         // cudaDeviceSynchronize();

//         // sleep(60);

//         // cudaMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, cudaMemcpyDeviceToHost);
//         // cudaMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, cudaMemcpyDeviceToHost);
//         // cudaMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, cudaMemcpyDeviceToHost);

//         // cudaFree(Ginfo);
//         // cudaFree(GfragmentInfo);
//         // cudaFree(GresidueInfo);
//         // cudaFree(GatomInfo);
//         // cudaFree(Ggrid);
//         // cudaFree(Gff);
//         // cudaFree(GmoveArray);

//     }



// }

